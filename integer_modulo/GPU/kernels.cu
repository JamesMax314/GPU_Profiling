
#include <hip/hip_runtime.h>
// Addition
__global__ void add(int n, int *a, int *b, int *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) c[i] = a[i] + b[i];
}

// Classic modulo
__global__ void modulo(int n, int *a, int *b, int *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) c[i] = a[i] % b[i];
}